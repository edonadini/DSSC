
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <math.h>

#define N (2048*2048)
#define THREAD_PER_BLOCK 512

__global__ void add( int *a, int *b, int *c ) {
       int index=threadIdx.x+blockIdx.x*blockDim.x;
       c[index] = a[index]+b[index];
}

void random_ints(int *p, int n) {
	int i;
	for(i=0; i<n; i++) {
		p[i]=rand();
	}
}

int main( void ) {
    int *a, *b, *c, *d;               // host copies of a, b, c
    int *dev_a, *dev_b, *dev_c;   // device copies of a, b, c
    int size = N * sizeof( int ); // we need space for N   									// integers
    int i;

    // allocate device copies of a, b, c
    hipMalloc( (void**)&dev_a, size );
    hipMalloc( (void**)&dev_b, size );
    hipMalloc( (void**)&dev_c, size );

    a = (int*)malloc( size ); 
    b = (int*)malloc( size );
    c = (int*)malloc( size );
    d = (int*)malloc( size );

    random_ints( a, N ); 
    random_ints( b, N );
    // copy inputs to device
   hipMemcpy( dev_a, a, size, hipMemcpyHostToDevice );
   hipMemcpy( dev_b, b, size, hipMemcpyHostToDevice );

    // launch an add() kernel with N threads
    add<<< N/THREAD_PER_BLOCK, THREAD_PER_BLOCK >>>( dev_a, dev_b, dev_c );

    // copy device result back to host copy of c
   hipMemcpy( c, dev_c, size,   hipMemcpyDeviceToHost );

    for(i=0; i<N; i++) {
	d[i]=a[i]+b[i];
	if(d[i]!=c[i]) {
	   printf("error: expected %d, got %d!\n",c[i], d[i]);
	   break;
	}
    }
    if(i==N) {
	   printf("correct!\n");
    }    

 
    free( a ); free( b ); free( c );
    hipFree( dev_a );
    hipFree( dev_b );
    hipFree( dev_c );
    return 0;
}
