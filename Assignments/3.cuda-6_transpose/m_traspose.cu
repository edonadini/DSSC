#include <stdio.h>
#include<stdlib.h>

#define nths 1024
#define n 8192

_global_ void mat_transp(double* m_in, double *m_out){
j=blockIdx.x;
i=threadIdx.x;
while(i<n){
m_out(i*n+j)=m_in(j*n+i);
i+=blockDim.x;}
}

_global_ void mat_fast_transp(){

}

int main(&argc, &argv[]){

double *mat_in_h, *mat_out_h, *mat_in_d, *mat_out_d;
int size=n*n*sizeof(double);
int nblock=(n*n)/nths;

mat_in_h=(double*)malloc(size);
mat_out_h=(double*)malloc(size);

hipMalloc(&mat_in_d,size);
hipMalloc(&mat_out_d,size);

//inizialize the matrix
for(int i=0;i<(n*n);i++) mat_in_h[i]=(double) i;

//move data from CPU to GPU
hipMemcpy( mat_in_d, mat_in_h, size, hipMemcpyHostToDevice);

//run a kernel
mat_transp<<<nblock,nths>>>( mat_in_d, mat_out_d);
hipMemcpy( mat_out_h, mat_out_d, size, hipMemcpyDeviceToDevice);

free(mat_in_h);
free(mat_out_h);
hipFree(mat_in_d);
hipFree(mat_out_h);
return 0;
}
