
#include <hip/hip_runtime.h>
#include <stdio.h>
#include<stdlib.h>

#define nths 1024
#define n 8192 //size of the matrix
#define dim 32

typedef void(*kernelFunc)(double*,double*,int);

__global__ void mat_transp(double* m_in, double *m_out, int siz){
int row=blockIdx.x*blockDim.x+threadIdx.x;
int col=blockIdx.y*blockDim.y+threadIdx.y;
m_out[col*siz+row]=m_in[row*siz+col];
}

__global__ void mat_fast_transp(double *m_in, double *m_out, int siz){
__shared__ double tile[dim][dim];
int col=blockIdx.x*blockDim.x +threadIdx.x;
int row=blockIdx.y*blockDim.y +threadIdx.y;
tile[threadIdx.x][threadIdx.y]=m_in[row*siz+col];
__syncthreads();
m_out[col*siz+row]=tile[threadIdx.x][threadIdx.y];
}

int correctness(double *m_in,double *m_out, int siz){
for (int i=0;i<siz; ++i)
	for(int j=0;j<siz; ++j)
		if(m_out[i*siz+j]!=m_in[j*siz+i])
			return 0;
return 1;
}

int TestCuda(kernelFunc kernel,const char *kernelName, int block_x, int block_y){
int size= n*n*sizeof(double);
double *mat_in_h, *mat_out_h, *mat_in_d, *mat_out_d;

dim3 block(block_x, block_y);
dim3 grid(n/block.x, n/block.y);

mat_in_h=(double*)malloc(size);
mat_out_h=(double*)malloc(size);

hipMalloc((void **)&mat_in_d,size);
hipMalloc((void **)&mat_out_d,size);

//inizialize the matrix
for(int i=0;i<(n*n);i++) mat_in_h[i]= i;

//move data from CPU to GPU
hipMemcpy( mat_in_d, mat_in_h, size, hipMemcpyHostToDevice);

//timing
hipEvent_t start, stop;
hipEventCreate(&start);
hipEventCreate(&stop);

hipEventRecord(start);
//run a kernel
kernel<<<grid,block>>>( mat_in_d, mat_out_d, dim);
hipEventRecord(stop);
hipEventSynchronize(stop);
//move data from GPU to CPU
hipMemcpy( mat_out_h, mat_out_d, size, hipMemcpyDeviceToHost);

//verify the correctness
printf("%s: %s\n",kernelName,correctness(mat_in_h,mat_out_h,size)? "Correct":"Fail");

float milliseconds=0;
hipEventElapsedTime(&milliseconds, start, stop);
printf("Time in milliseconds: %f\n", milliseconds);
printf("Bandwidth: %f GB/s\n", 2*size/milliseconds/1e6);
printf("-------------------\n");

//Cleanup
free(mat_in_h);
free(mat_out_h);
hipFree(mat_in_d);
hipFree(mat_out_h);
hipEventDestroy(start);
hipEventDestroy(stop);
return 0;
}

int main(int argc, char*argv[]){
TestCuda(&mat_transp,"Naive Transpose threads per block 64", 8,8);
TestCuda(&mat_transp,"Naive Transpose threads per block 512", 16,32);
TestCuda(&mat_transp, "Naive Transpose threads per block 512",32,16);
TestCuda(&mat_transp, "Naive Transpose threads per block 1024", 32,32);
TestCuda(&mat_fast_transp, "Optimized Transpose threads per block 64", 8,8);
TestCuda(&mat_fast_transp,"Optimized Transpose threads per blocks 512", 16,32);
TestCuda(&mat_fast_transp,"Optimized Transpose threads per block 512",32,16);
TestCuda(&mat_fast_transp,"Optimized Transpose threads per blocks 1024",32,32);

return 0;
}
